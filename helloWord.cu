
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello World from GPU!\n");
}

int main() {
    // Launch kernel
    helloFromGPU<<<1, 1>>>();

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    return 0;
}
